//============================================================================
// Name        : inverse.cpp
// Author      : Erman Gurses
//============================================================================

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

#define lookup2D(ptr,x,y,matY) (ptr)[ (x) * (matY) + (y)]

#define lookup2DK(ptr,threadId,x,y,matY) (ptr)[ (threadId)*(matY)*(matY)+ (x) * (matY) + (y)]

#define lookup3D(ptr,i,j,k,Nx,Ny,Nz) &(ptr)[( (Nx) * (Ny) * (k)   +           \
                                              (Nx) * (j ) + (i) ) * matY*matY ]

#define lookup5D(ptr,i,j,k,Nx,Ny,Nz,x,y,matY) (ptr)[((Nx) * (Ny)    * (k)   +            \
                                                     (Nx) *  (j)    + (i))  * matSize +  \
                                                      (x) *  (matY) + (y)]

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
   if (code != hipSuccess){
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__device__ int getGlobalIdx_3D_3D(){ 

int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;

int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z   ) + 
                         (threadIdx.z * (blockDim.x * blockDim.y)) + 
                         (threadIdx.y * blockDim.x) + threadIdx.x;

  return threadId; 
}

__global__ void inverse(double *A, double *B, double *C, int matY){

int blockId = blockIdx.x + blockIdx.y * gridDim.x + 
               gridDim.x * gridDim.y * blockIdx.z;

int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + 
                         (threadIdx.z * (blockDim.x * blockDim.y)) + 
                         (threadIdx.y * blockDim.x) + threadIdx.x;  


//printf("threadId %d\n",threadId);
  
 __shared__ double temp[3];
      
  for (int j = 0; j < matY; j++){
    for (int i = 0; i < matY; i++){
      lookup2DK(B,threadId,j,i,matY) = 0,0;
      lookup2DK(C,threadId,j,i,matY) =lookup2DK(A,threadId,j,i,matY);
    }
    lookup2DK(B,threadId,j,j,matY) = 1.0;
  }

  for (int k = 0; k < matY-1; k++){
    for (int j = 0; j < matY; j++){
      lookup2DK(B,threadId,k,j,matY) = lookup2DK(B,threadId,k,j,matY) /
			               lookup2DK(C,threadId,k,k,matY);
    }
    for (int j = 0; j < matY; j++){
      temp[j] = lookup2DK(C,threadId,k,k,matY);
    }
    for (int j = 0; j < matY; j++){
      lookup2DK(C,threadId,k,j,matY) = lookup2DK(C,threadId,k,j,matY) / temp[j];
    }
    for (int i = k + 1; i < matY; i++){
      for (int l = 0; l < matY; l++){
        lookup2DK(B,threadId,i,l,matY) = 
                  lookup2DK(B,threadId,i,l,matY) -
	          lookup2DK(C,threadId,i,k,matY) * 
                  lookup2DK(B,threadId,k,l,matY);

         temp[l] = lookup2DK(C,threadId,i,l,matY) -
                   lookup2DK(C,threadId,i,k,matY) * 
                   lookup2DK(C,threadId,k,l,matY);
      }//l
      for (int l = 0; l < matY; l++){
        lookup2DK(C,threadId,i,l,matY)   = temp[l];
      }//l
    } //i
  }//k
  for (int i = 0; i < matY; i++){
    lookup2DK(B,threadId,matY-1,i,matY) = lookup2DK(B,threadId,matY-1,i,matY) /
			                  lookup2DK(B,threadId,matY-1,matY-1,matY);

    lookup2DK(C,threadId,matY-1,i,matY) = lookup2DK(C,threadId,matY-1,i,matY) /
			                  lookup2DK(C,threadId,matY-1,matY-1,matY);
  }

  for (int k = matY-1; k >= 1; k--){
    for (int i = 0; i <= k-1; i++){
      for (int l = 0; l < matY; l++){
        lookup2DK(B,threadId,i,l,matY) = 
                  lookup2DK(B,threadId,i,l,matY) - 
                  lookup2DK(C,threadId,i,k,matY) * 
                  lookup2DK(B,threadId,k,l,matY);

        temp[l] = lookup2DK(C,threadId,i,l,matY) -
                  lookup2DK(C,threadId,i,k,matY) *
	          lookup2DK(C,threadId,k,l,matY);
      }//l
      for (int l = 0; l < matY; l++){
        lookup2DK(C,threadId,i,l,matY) = temp[l];
      }//l
    }//i
  }//k 
}

int main() {
  int k, j, i;
  int Nx = 4;
  int Ny = 4;
  int Nz = 4;
  int matX= 3;
  int matY= 3;
  int matSize = matX * matY;
  int requiredSpaceBlock = sizeof(double) * matSize;
  int requiredSpace = requiredSpaceBlock * Nx * Ny * Nz;
  double *B, *M, *C, *B_Device, *M_Device, *C_Device;

  M = (double*) malloc(requiredSpace);
  B = (double*) malloc(requiredSpace);  
  C = (double*) malloc(requiredSpace);

  for(k = 0; k < Nz; k++){
    for(j = 0; j < Ny; j++){
      for(i = 0; i < Nx; i++){
        lookup5D(M,i,j,k,Nx,Ny,Nz,0,0,matY) =  1;
        lookup5D(M,i,j,k,Nx,Ny,Nz,0,1,matY) = -1;
        lookup5D(M,i,j,k,Nx,Ny,Nz,0,2,matY) = -1;
        lookup5D(M,i,j,k,Nx,Ny,Nz,1,0,matY) = -1;
        lookup5D(M,i,j,k,Nx,Ny,Nz,1,1,matY) =  2;
        lookup5D(M,i,j,k,Nx,Ny,Nz,1,2,matY) =  3;
        lookup5D(M,i,j,k,Nx,Ny,Nz,2,0,matY) =  1;
        lookup5D(M,i,j,k,Nx,Ny,Nz,2,1,matY) =  1;   
        lookup5D(M,i,j,k,Nx,Ny,Nz,2,2,matY) =  4;
      }
    }
  } 
   
  gpuErrchk(hipMalloc((void**)&B_Device,requiredSpace));
  gpuErrchk(hipMalloc((void**)&M_Device,requiredSpace));
  gpuErrchk(hipMalloc((void**)&C_Device,requiredSpace));
  
  gpuErrchk(hipMemcpy(B_Device,B,requiredSpace,hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(M_Device,M,requiredSpace,hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(C_Device,C,requiredSpace,hipMemcpyHostToDevice));
              
  int blockX = 2; 
  int blockY = 2; 
  int blockZ = 2; 

  int gridX = ceil(Nx/blockX); 
  int gridY = ceil(Ny/blockY);
  int gridZ = ceil(Nz/blockZ);
   
  dim3 blockSize(blockX,blockY,blockZ);
  dim3 gridSize(gridX,gridY,gridZ);
 
  // Kernel Call       
  inverse<<<gridSize,blockSize>>>(M_Device,B_Device,C_Device,matY);
 
  gpuErrchk( hipPeekAtLastError() ); 
  gpuErrchk(hipDeviceSynchronize());
      
  gpuErrchk(hipMemcpy(B,B_Device,requiredSpace,hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(M,M_Device,requiredSpace,hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(C,C_Device,requiredSpace,hipMemcpyDeviceToHost));
  

  for(int k = 0; k < Nz; k++){
    for(int j = 0; j < Ny; j++){
      for(int i = 0; i < Nx; i++){
        printf("%9f  ",lookup5D(B,i,j,k,Nx,Ny,Nz,0,0,matY));
        printf("%9f  ",lookup5D(B,i,j,k,Nx,Ny,Nz,0,1,matY));
        printf("%9f  ",lookup5D(B,i,j,k,Nx,Ny,Nz,0,2,matY));
        printf("\n");

        printf("%9f  ",lookup5D(B,i,j,k,Nx,Ny,Nz,1,0,matY));
        printf("%9f  ",lookup5D(B,i,j,k,Nx,Ny,Nz,1,1,matY));
        printf("%9f  ",lookup5D(B,i,j,k,Nx,Ny,Nz,1,2,matY));
        printf("\n");

        printf("%9f  ",lookup5D(B,i,j,k,Nx,Ny,Nz,2,0,matY));
        printf("%9f  ",lookup5D(B,i,j,k,Nx,Ny,Nz,2,1,matY));
        printf("%9f  ",lookup5D(B,i,j,k,Nx,Ny,Nz,2,2,matY));
        printf("\n\n");
      }
    }
  }

 return 0;
}

